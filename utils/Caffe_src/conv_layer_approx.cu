#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/conv_layer_approx.hpp"
//#include "caffe/util/approx_mult.hpp"




#define P 8 
#define MAX 1<<(15-P)

namespace caffe{

__device__   int leadingBitPosition_conv(int val)
  {
    unsigned n = 0, x;
    x = val;
    if (x <= 0x0000ffff) n += 16, x <<= 16;
    if (x <= 0x00ffffff) n += 8, x <<= 8;
    if (x <= 0x0fffffff) n += 4, x <<= 4;
    if (x <= 0x3fffffff) n += 2, x <<= 2;
    if (x <= 0x7fffffff) n++;
    return 31 - n;
  }


__device__ int ROBA_conv(int x, int y){
	unsigned short x_abs, y_abs;
	int p;
	unsigned int p_abs;
	char sgn_x = x > 0 ? 0 : 1;
	char sgn_y = y > 0 ? 0 : 1;
	char kx, ky;
	x_abs = sgn_x ? -(x)  : x;
	y_abs = sgn_y ? -(y) : y;

	uint16_t x_round, y_round;
	char zero = x_abs != 0 & y_abs != 0; 

	kx = leadingBitPosition_conv(x_abs);
	x_round = (x_abs >= 3*(1 << (kx-1))) ? 1 << (kx+1) : 1 << kx;
	x_round = (x_abs == 3) ? 3 : x_round;

	ky = leadingBitPosition_conv(y_abs);
	y_round = (y_abs >= 3*(1 << (ky-1))) ? 1 << (ky+1) : 1 << ky;
	y_round = (y_abs == 3) ? 2 : y_round;


	p_abs =  (x_round * y_abs) +  (y_round * x_abs) - (y_round * x_round);

	p = sgn_x ^ sgn_y ? -p_abs : p_abs;

	return p*zero;
	

}


template <typename Dtype>
__device__ Dtype mult_fixed_conv(const Dtype *a, const Dtype *b)
{
  int x, y;
  int z;
  // Cutting off in quantization
  x = (short)(*a * (1 << P));
  y = (short)(*b * (1 << P));
  x = *a >= MAX ? (1<<15)-1 : x;
  x = *a <= -MAX ? -(1<<15) : x;
  y = *b >= MAX ? (1<<15)-1 : y;
  y = *b <= -MAX ? -(1<<15) : y;
  z = ROBA_conv(x,y); 
  return ((Dtype)z / (1 << 2 * P));
 //return *a * *b;
}


  template <typename Dtype>
__global__ void ConvForward(const int nthreads,
		const Dtype* const bottom_data, const int num, const int channels,
		const int height, const int width,const int conved_height,
		const int conved_width,const int kernel_h, const int kernel_w, const int kernel_n,
		const int stride_h, const int stride_w, const int pad_h, const int pad_w,
		Dtype* const top_data,const Dtype* const weight,const Dtype* const bias,const bool bias_term_) {
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int pw = index % conved_width;
    const int ph = (index / conved_width) % conved_height;
    // kernel_n denotes the number of filters which is equal to the number of channels 
    const int c = (index / conved_width / conved_height) % kernel_n;
		const int n = index / conved_width / conved_height / kernel_n;
    
    int hstart = ph * stride_h - pad_h;
		int wstart = pw * stride_w - pad_w;
		int hend = min(hstart + kernel_h, height + pad_h);
		int wend = min(wstart + kernel_w, width + pad_w);
//		const int pool_size = (hend - hstart) * (wend - wstart);
		hstart = max(hstart, 0);
		wstart = max(wstart, 0);
		hend = min(hend, height);
		wend = min(wend, width);
    Dtype aveval = 0;
    
//		if (index==1) {
//			printf("pw%d ph%d c%d n%d \n",pw,ph,c,n);
//			printf("hstart%d wstart%d hend%d wend%d \n",hstart,wstart,hend,wend);
//		}
    for(int ch = 0; ch < channels; ++ch){
      int khstart=hend<kernel_h?kernel_h-hend:0;
      int kwstart=wend<kernel_w?kernel_w-wend:0;
      const Dtype*  bottom_slice = bottom_data + (n * channels + ch) * height * width;
      const Dtype*  weight_slice = weight + (c * channels + ch) * kernel_h * kernel_w;
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {

          //aveval += bottom_slice[h * width + w]*weight_slice[(khstart+h-hstart) * kernel_w + (kwstart+w-wstart)];
          aveval += mult_fixed_conv(&bottom_slice[h * width + w],&weight_slice[(khstart+h-hstart) * kernel_w + (kwstart+w-wstart)]);

        }
      }
    }
		if(bias_term_) {  
			aveval+=bias[c];
		}
		top_data[index] = aveval;
	}
}




  // This code needs to be modified 
template <typename Dtype>
void ConvolutionApproxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    //	std::cout << "fp" << std::endl;
	const Dtype* weight = this->blobs_[0]->gpu_data();
	int* kernel_shape_data = this->kernel_shape_.mutable_cpu_data();
	int* stride_data = this->stride_.mutable_cpu_data();
	int* pad_data = this->pad_.mutable_cpu_data();

	for (int i = 0; i < bottom.size(); ++i) {
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype* top_data = top[i]->mutable_gpu_data();
		const int count = top[i]->count();
		vector<int> shape_ = bottom[i]->shape();
		const int channels_ = shape_[1];
		const int height_ = shape_[2];
		const int width_ = shape_[3];

    
    // number_of_outputs 
    vector<int> weight_shape_ = top[i]->shape();
    const int kernel_n_ = weight_shape_[1];


    const int kernel_h_ = kernel_shape_data[0];
		const int kernel_w_ = kernel_shape_data[1];
		const int stride_h_ = stride_data[0];
		const int stride_w_ = stride_data[1];
		const int pad_h_ = pad_data[0];
		const int pad_w_ = pad_data[1];

		const int conved_height = this->output_shape_[0];
		const int conved_weight = this->output_shape_[1];
    
		const bool bias_term_ = this->bias_term_;

		if (bias_term_) {
			const Dtype* const bias = this->blobs_[1]->gpu_data();
			ConvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, bottom[i]->num(), channels_,
					height_, width_,conved_height,conved_weight,kernel_h_,
					kernel_w_, kernel_n_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,weight,bias,bias_term_);
		} else {
			ConvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, bottom[i]->num(), channels_,
					height_, width_,conved_height,conved_weight,kernel_h_,
					kernel_w_, kernel_n_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,weight,0,bias_term_);
		}
	}
}

template <typename Dtype>
void ConvolutionApproxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionApproxLayer);

}  // namespace caffe
