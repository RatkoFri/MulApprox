#include "hip/hip_runtime.h"


#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_layer_approx.hpp"
#include "caffe/util/math_functions.hpp"

#define P 8 

#define P 8
#define MAX 1<<(15-P)

namespace caffe{

__device__   int leadingBitPosition_fc(int val)
  {
    unsigned n = 0, x;
    x = val;
    if (x <= 0x0000ffff) n += 16, x <<= 16;
    if (x <= 0x00ffffff) n += 8, x <<= 8;
    if (x <= 0x0fffffff) n += 4, x <<= 4;
    if (x <= 0x3fffffff) n += 2, x <<= 2;
    if (x <= 0x7fffffff) n++;
    return 31 - n;
  }

__device__ int ROBA_fc(int x, int y){
	unsigned short x_abs, y_abs;
	int p;
	unsigned int p_abs;
	char sgn_x = x > 0 ? 0 : 1;
	char sgn_y = y > 0 ? 0 : 1;
	char kx, ky;
	x_abs = sgn_x ? -(x)  : x;
	y_abs = sgn_y ? -(y) : y;

	uint16_t x_round, y_round;
	char zero = x_abs != 0 & y_abs != 0; 

	kx = leadingBitPosition_fc(x_abs);
	x_round = (x_abs >= 3*(1 << (kx-1))) ? 1 << (kx+1) : 1 << kx;
	x_round = (x_abs == 3) ? 3 : x_round;

	ky = leadingBitPosition_fc(y_abs);
	y_round = (y_abs >= 3*(1 << (ky-1))) ? 1 << (ky+1) : 1 << ky;
	y_round = (y_abs == 3) ? 2 : y_round;


	p_abs =  (x_round * y_abs) +  (y_round * x_abs) - (y_round * x_round);

	p = sgn_x ^ sgn_y ? -p_abs : p_abs;

	return p*zero;
	

}


template <typename Dtype>
__device__ Dtype mult_fixed_fc(const Dtype *a, const Dtype *b)
{
  int x, y;
  int z;
  // Cutting off in quantization
  x = (short)(*a * (1 << P));
  y = (short)(*b * (1 << P));
  x = *a >= MAX ? (1<<15)-1 : x;
  x = *a <= -MAX ? -(1<<15) : x;
  y = *b >= MAX ? (1<<15)-1 : y;
  y = *b <= -MAX ? -(1<<15) : y;
	z = ROBA(x,y); 
  return ((Dtype)z / (1 << 2 * P));
 //return *a * *b;
}

  template <typename Dtype>
__global__ void FCCForward(const int nthreads,
		const Dtype* bottom_data, const Dtype*  weight,
    Dtype* top_data, int M, int N, int K, const Dtype* bias,
    const int bias_term_, const Dtype* const bias_multiplier) {
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int pw = index % N;
    const int ph = index / N;

    Dtype aveval = 0;
    
//		if (index==1) {
//			printf("pw%d ph%d c%d n%d \n",pw,ph,c,n);
//			printf("hstart%d wstart%d hend%d wend%d \n",hstart,wstart,hend,wend);
//		}


   
  
    for(int pk = 0; pk < K; pk++){

      // aveval += bottom_data[ph*K + pk]*weight[pk + pw*K];
      // aveval += mult_fixed((double)bottom_data[ph*K + pk],(double)weight[pk + pw*K]);
      aveval += mult_fixed_fc(bottom_data+ph*K + pk,weight + pk + pw*K);
    }

     // Bias multiplier needs to be checked, I have a bad feeling that  something isn't working like it should. Still, we managed to 
     // create inner product. At the end filter were in shape of N*K not K*N
		 if(bias_term_) {  
		 	aveval+=bias[pw]*bias_multiplier[ph];
	  }
		top_data[index] = aveval;
	}
}

  

template <typename Dtype>
void InnerProductApproxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();

  if (bias_term_) {
    const Dtype* const bias = this->blobs_[1]->gpu_data();
    FCCForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,bottom_data, weight, top_data, M_, N_, K_,bias,bias_term_,bias_multiplier_.gpu_data());
  } else {
    FCCForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,bottom_data, weight, top_data, M_, N_, K_,0,bias_term_,bias_multiplier_.gpu_data());
  }
  //  printf("Print %d \n", bottom.size());

  // for (int i = 0; i < bottom.size(); ++i) {
  //   const Dtype* bottom_data = bottom[i]->gpu_data();
	// 	Dtype* top_data = top[i]->mutable_gpu_data();
	// 	const int count = top[i]->count();
  //   if (bias_term_) {
  //       const Dtype* const bias = this->blobs_[1]->gpu_data();
  //       FCCForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //           count,bottom_data, weight, top_data, M_, N_, K_,bias,bias_term_);
  //     } else {
  //       FCCForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //           count,bottom_data, weight, top_data, M_, N_, K_,0,bias_term_);
  //     }
  // }


}

template <typename Dtype>
void InnerProductApproxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          K_, N_, M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          M_, K_, N_,
          (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          M_, K_, N_,
         (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductApproxLayer);

}  // namespace caffe
